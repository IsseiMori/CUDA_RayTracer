#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "camera.h"
#include "hitable_list.h"
#include "material.h"

using namespace std;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " <<
		file << ":" << line << "'" << func << "/n";
		hipDeviceReset();
		exit(99);
	}
}


__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0,1.0,1.0);
	for (int i = 0; i < 50; i++) {
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
			else {
				return vec3(0.0,0.0,0.0);
			}
		}
		else {
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f*(unit_direction.y()+1.0f);
			vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0)+t*vec3(0.5,0.7,1.0);
			return cur_attenuation * c;
		}
	}
	// reached max recursion
	return vec3(0.0, 0.0, 0.0);
	
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1004, 0, 0, rand_state);
    }
}

/* Initialize rand function so that each thread will have guaranteed distinct random numbrs*/
__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;

	int pixel_index = j*max_x+i;
	hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
	
	int pixel_index = j*max_x+i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for (int s = 0; s < ns; s++) {
    	float u = float(i+hiprand_uniform(&local_rand_state))/float(max_x);
	    float v = float(j+hiprand_uniform(&local_rand_state))/float(max_y);
	    ray r = (*cam)->get_ray(u,v,&local_rand_state);
	    col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
    
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -5; a < 5; a++) {
            for(int b = -5; b < 5; b++) {
                float choose_mat = RND;
                vec3 center(1.5f*a+RND,0.2,1.5f*b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, 10*10+1+3);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
	for (int i = 0; i < 10*10+1+3; i++) {
		delete ((sphere *)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}

int main() {
	int image_width = 640;
	int image_height = 480;
	int ns = 50;
	int tx = 8;
	int ty = 8;

	std::ofstream outfile;
	outfile.open("outfile.ppm");
	outfile << "P3\n" << image_width << " " << image_height << "\n255\n";
	
	int num_pixels = image_width*image_height;
	size_t fb_size = num_pixels*sizeof(vec3);

	vec3 *fb;
	checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

	// allocate random generator
	hiprandState *d_rand_state;
	checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels*sizeof(hiprandState)));
	hiprandState *d_rand_state2;
	checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1*sizeof(hiprandState)));

	rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

	hitable **d_list;
	int num_hitables = 10*10+1+3;
	checkCudaErrors(hipMalloc((void**)&d_list, num_hitables*sizeof(hitable*)));
	hitable **d_world;
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable_list*)));
	camera **d_camera;
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
	create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();

	dim3 blocks(image_width/tx+1, image_height/ty+1);
	dim3 threads(tx, ty);
	render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	render<<<blocks, threads>>>(fb, image_width, image_height, ns, d_camera, d_world, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();
	double timer_seconds = ((double)(stop-start) / CLOCKS_PER_SEC);
	cout << "Renderd in " << timer_seconds << " [s]." << endl;

	

	for (int j = image_height-1; j >= 0; j--) {
		for (int i = 0; i < image_width; i++) {
			size_t pixel_index = j*image_width+i;
			int ir = int(255.99*fb[pixel_index].x());
			int ig = int(255.99*fb[pixel_index].y());
			int ib = int(255.99*fb[pixel_index].z());

			outfile << ir << " " << ig << " " << ib << "\n";
		}
	}

	outfile.close();

	checkCudaErrors(hipDeviceSynchronize());
	free_world<<<1,1>>>(d_list, d_world, d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(fb));

	hipDeviceReset();
}