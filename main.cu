#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <fstream>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"

using namespace std;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " <<
		file << ":" << line << "'" << func << "/n";
		hipDeviceReset();
		exit(99);
	}
}


__device__ vec3 color(const ray& r, hitable **world) {
	hit_record rec;
	if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
		return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f);
	}
	else {
		vec3 unit_direction(r.direction());
		float t = 0.5f*(unit_direction.y()+1.0f);
		return (1.0f-t)*vec3(1.0, 1.0, 1.0)+t*vec3(0.5,0.7,1.0);
	}
}

__global__ void render(vec3 *fb, int max_x, int max_y, 
					   vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin,
					   hitable **world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j*max_x+i;
    float u = float(i)/float(max_x);
    float v = float(j)/float(max_y);
    ray r(origin, lower_left_corner+u*horizontal+v*vertical);
    fb[pixel_index] = color(r, world);
}

__global__ void create_world(hitable **d_list, hitable **d_world) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		*(d_list) = new sphere(vec3(0, 0, -1), (0.5));
		*(d_list+1) = new sphere(vec3(0, -100.5, -1), 100);
		*d_world = new hitable_list(d_list, 2);
	}
}

__global__ void free_world(hitable **d_list, hitable_list **d_world) {
	delete *(d_list);
	delete *(d_list+1);
	delete *(d_world);
}

int main() {
	const auto aspect_ratio = 16.0 / 9.0;
	const int image_width = 384;
	const int image_height = static_cast<int>(image_width / aspect_ratio);

	std::ofstream outfile;
	outfile.open("outfile.ppm");
	outfile << "P3\n" << image_width << " " << image_height << "\n255\n";
	
	auto viewport_height = 2.0f;
	auto viewpor_width = aspect_ratio * viewport_height;
	auto focal_length = 1.0f;

	auto origin = vec3(0, 0, 0);
	auto horizontal = vec3(viewpor_width, 0, 0);
	auto vertical = vec3(0, viewport_height, 0);
	auto lower_left_corner = origin - horizontal/2 - vertical/2 - vec3(0, 0, focal_length);

	int num_pixels = image_width*image_height;
	size_t fb_size = num_pixels*sizeof(vec3);

	vec3 *fb;
	checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

	hitable **d_list;
	checkCudaErrors(hipMalloc((void**)&d_list, 2*sizeof(hitable*)));
	hitable **d_world;
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable_list*)));
	create_world<<<1,1>>>(d_list, d_world);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();

	int tx = 16;
	int ty = 16;

	dim3 blocks(image_width/tx+1, image_height/ty+1);
	dim3 threads(tx, ty);

	render<<<blocks, threads>>>(fb, image_width, image_height, lower_left_corner, 
								horizontal, vertical, origin, d_world);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();
	double timer_seconds = ((double)(stop-start) / CLOCKS_PER_SEC);
	cout << "Renderd in " << timer_seconds << " [s]." << endl;

	

	for (int j = image_height-1; j >= 0; j--) {
		for (int i = 0; i < image_width; i++) {
			size_t pixel_index = j*image_width+i;
			int ir = int(255.99*fb[pixel_index].x());
			int ig = int(255.99*fb[pixel_index].y());
			int ib = int(255.99*fb[pixel_index].z());

			outfile << ir << " " << ig << " " << ib << "\n";
		}
	}

	outfile.close();
	checkCudaErrors(hipFree(fb));
}