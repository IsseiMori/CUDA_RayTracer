#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "camera.h"
#include "hitable_list.h"
#include "material.h"

using namespace std;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at " <<
		file << ":" << line << "'" << func << "/n";
		hipDeviceReset();
		exit(99);
	}
}


__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0,1.0,1.0);
	for (int i = 0; i < 50; i++) {
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
			else {
				return vec3(0.0,0.0,0.0);
			}
		}
		else {
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f*(unit_direction.y()+1.0f);
			vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0)+t*vec3(0.5,0.7,1.0);
			return cur_attenuation * c;
		}
	}
	// reached max recursion
	return vec3(0.0, 0.0, 0.0);
	
}

/* Initialize rand function so that each thread will have guaranteed distinct random numbrs*/
__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;

	int pixel_index = j*max_x+i;
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
	
	int pixel_index = j*max_x+i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for (int s = 0; s < ns; s++) {
    	float u = float(i+hiprand_uniform(&local_rand_state))/float(max_x);
	    float v = float(j+hiprand_uniform(&local_rand_state))/float(max_y);
	    ray r = (*cam)->get_ray(u,v,&local_rand_state);
	    col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
    
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		d_list[0] = new sphere(vec3(0,0,-1), 0.5,
                               new lambertian(vec3(0.1, 0.2, 0.5)));
        d_list[1] = new sphere(vec3(0,-100.5,-1), 100,
                               new lambertian(vec3(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(vec3(1,0,-1), 0.5,
                               new metal(vec3(0.8, 0.6, 0.2), 0.0));
        d_list[3] = new sphere(vec3(-1,0,-1), 0.5,
                               new dielectric(1.5));
        d_list[4] = new sphere(vec3(-1,0,-1), -0.45,
                               new dielectric(1.5));
		*d_world = new hitable_list(d_list, 5);
		vec3 lookfrom(3,3,2);
        vec3 lookat(0,0,-1);
        float dist_to_focus = (lookfrom-lookat).length();
        float aperture = 2.0;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 20.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
	}
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
	for (int i = 0; i < 4; i++) {
		delete ((sphere *)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}

int main() {
	int image_width = 600;
	int image_height = 300;
	int ns = 100;
	int tx = 8;
	int ty = 8;

	std::ofstream outfile;
	outfile.open("outfile.ppm");
	outfile << "P3\n" << image_width << " " << image_height << "\n255\n";
	
	int num_pixels = image_width*image_height;
	size_t fb_size = num_pixels*sizeof(vec3);

	vec3 *fb;
	checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

	// allocate random generator
	hiprandState *d_rand_state;
	checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels*sizeof(hiprandState)));

	hitable **d_list;
	checkCudaErrors(hipMalloc((void**)&d_list, 4*sizeof(hitable*)));
	hitable **d_world;
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable_list*)));
	camera **d_camera;
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
	create_world<<<1,1>>>(d_list, d_world, d_camera, image_width, image_height);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();

	dim3 blocks(image_width/tx+1, image_height/ty+1);
	dim3 threads(tx, ty);

	render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	render<<<blocks, threads>>>(fb, image_width, image_height, ns, d_camera, d_world, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();
	double timer_seconds = ((double)(stop-start) / CLOCKS_PER_SEC);
	cout << "Renderd in " << timer_seconds << " [s]." << endl;

	

	for (int j = image_height-1; j >= 0; j--) {
		for (int i = 0; i < image_width; i++) {
			size_t pixel_index = j*image_width+i;
			int ir = int(255.99*fb[pixel_index].x());
			int ig = int(255.99*fb[pixel_index].y());
			int ib = int(255.99*fb[pixel_index].z());

			outfile << ir << " " << ig << " " << ib << "\n";
		}
	}

	outfile.close();

	checkCudaErrors(hipDeviceSynchronize());
	free_world<<<1,1>>>(d_list, d_world, d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(fb));

	hipDeviceReset();
}